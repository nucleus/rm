#include "hip/hip_runtime.h"
/*
 * CUDA source file for ImplicitSurface class.
 */

#define check(expr, msg) { \
	hipError_t err; \
	err = expr; \
	if (err != hipSuccess) { std::cerr << "CUDA ERROR (" << hipGetErrorString(err) << "): " << msg << std::endl; } \
	}
	
#define checkThrust(expr, msg) { \
	try { \
	expr; \
	} catch(thrust::system_error &e) { \
		std::cerr << "THRUST ERROR (" << e.what() << "): " << msg << std::endl; \
	} \
	}

#include <util/interface/Util.hpp>
#include <util/interface/ImplicitSurface.hpp>
#include <glm/interface/gtc/type_ptr.hpp>

#include <util/interface/cutil_math.h>

#include <thrust/device_ptr.h>
#include <thrust/copy.h>
#include <thrust/device_vector.h>

#include <limits.h>
#include <float.h>

// constant data
__constant__ float3 c_bboxMin;
__constant__ float3 c_bboxMax;

// textures
texture<float4, 3, hipReadModeElementType> tex_voxels;

inline __device__ glm::vec3 make_vec3(const float3& a) {
	return glm::vec3(a.x, a.y, a.z);
}

__device__
bool clipRayAgainstAABB(const glm::vec3& org, const glm::vec3& dir, float& tnear, float& tfar) {
	glm::vec3 T_1, T_2;
	tnear = -FLT_MAX;
	tfar = FLT_MAX;
	
	glm::vec3 min(make_vec3(c_bboxMin));
	glm::vec3 max(make_vec3(c_bboxMax));
	
	for (int i = 0; i < 3; i++){
		if (dir[i] == 0){
			if ((org[i] < min[i]) || (org[i] > max[i])) {
				return false;
			}
		} else {
			T_1[i] = (min[i] - org[i]) / dir[i];
			T_2[i] = (max[i] - org[i]) / dir[i];

			if (T_1[i] > T_2[i]) {
				glm::vec3 tmp = T_1;
				T_1 = T_2;
				T_2 = tmp;
			}
			if (T_1[i] > tnear) {
				tnear = T_1[i];
			}
			if (T_2[i] < tfar) {
				tfar = T_2[i];
			}
			if ( (tnear > tfar) || (tfar < 0.0f) || (fabsf(tfar - tnear) < 0.0001f) ) {
				return false;
			}
		}
	}
	
	return true;
}

inline __device__
void evaluateGrid(const glm::vec3& target, float& value, glm::vec3& normal) {
	glm::vec3 coords = (target - make_vec3(c_bboxMin)) / (make_vec3(c_bboxMax) - make_vec3(c_bboxMin));
	float4 interp = tex3D(tex_voxels, coords.x, coords.y, coords.z);
	value = interp.w;
	normal = glm::vec3(interp.x, interp.y, interp.z);
}

__global__
void raymarchKernel(glm::vec3 org, glm::vec3* d_rays, PointNormalPair* d_output, unsigned n, unsigned steps) {
	unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (idx > n) {
		return;
	}
	
	glm::vec3 dir = d_rays[idx];
	
	glm::vec3 hit, normal;
	float tnear, tfar;
	
	bool intersects = clipRayAgainstAABB(org, dir, tnear, tfar);
	if (!intersects) {
		hit = glm::vec3(0.0f);
		normal = glm::vec3(0.0f);
	} else {
		tnear += 0.00001f;

		// march ray through surface
		float step = (tfar - tnear) / steps;

		// set sign to unknown at the beginning
		char sign = 0;
		
		// find location where sign is defined first
		int i = 0;
		for (i = 1; i < steps-1; i++) { // -1 to not get too close to tfar
			glm::vec3 target = org + dir * (tnear + i * step);
			float implicitValue; glm::vec3 implicitNormal;
			
			evaluateGrid(target, implicitValue, implicitNormal);
			
			if (implicitValue < 0.0f) {
				sign = -1;
				break;
			} else if (implicitValue > 0.0f) {
				sign = 1;
				break;
			}
		}
		
		// trace until sign changes
		if (sign) {
			for(; i < steps-1; i++) {
				glm::vec3 target = org + dir * (tnear + i * step);
				float implicitValue; glm::vec3 implicitNormal;
				
				evaluateGrid(target, implicitValue, implicitNormal);
				
				if ((sign == 1 && implicitValue < 0.0f) || (sign == -1 && implicitValue > 0.0f)) { // ray entered surface
					float tStart = tnear + (i-1) * step;
					step = (tnear + i * step - tStart) / steps;
					
					for (int j = steps-1; j >= 0; j--) {
						target = org + dir * (tStart + j * step);
						
						evaluateGrid(target, implicitValue, normal);
						
						if ((sign == 1 && implicitValue > 0.0f) || (sign == -1 && implicitValue < 0.0f)) {
							hit = target;
							normal = implicitNormal;
							break;
						}
					}
					break;
				}
			}
		}
	}
	
	d_output[idx].first = hit;
 	d_output[idx].second = normal;
}

struct IsNotZero {
	__host__ __device__
	bool operator()(const PointNormalPair& a) {
		return !(a.first.x == 0.0f && a.first.y == 0.0f && a.first.z == 0.0f);
	}
};

void launchRaymarchKernel(const util::Grid3D& volume, const util::RayVector& rays, unsigned steps, PointNormalData& results) {
	// configure device
	check( hipDeviceSetCacheConfig( hipFuncCachePreferL1 ), "hipDeviceSetCacheConfig" );
	
	// copy bbox data to const
	check( hipMemcpyToSymbol(HIP_SYMBOL(c_bboxMin), glm::value_ptr(volume.bounds().min()), sizeof(float3)), "hipMemcpyToSymbol" );
	check( hipMemcpyToSymbol(HIP_SYMBOL(c_bboxMax), glm::value_ptr(volume.bounds().max()), sizeof(float3)), "hipMemcpyToSymbol" );
	
	// initialize grid textures
	hipArray* d_volumeArray;
	Point3i dims = volume.dimensions();
	const hipExtent volumeSize = make_hipExtent(dims.x, dims.y, dims.z);
	
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
	check( hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize), "hipMalloc3DArray" );
	
	hipMemcpy3DParms copyParams = {0};
	copyParams.srcPtr   = make_hipPitchedPtr((void*)volume.data(), volumeSize.width*sizeof(float)*4, volumeSize.width, volumeSize.height);
	copyParams.dstArray = d_volumeArray;
	copyParams.extent   = volumeSize;
	copyParams.kind     = hipMemcpyHostToDevice;
	check( hipMemcpy3D(&copyParams), "hipMemcpy3D" );

	tex_voxels.normalized = true;
	tex_voxels.filterMode = hipFilterModeLinear;
	tex_voxels.addressMode[0] = hipAddressModeClamp;
	tex_voxels.addressMode[1] = hipAddressModeClamp;
	tex_voxels.addressMode[2] = hipAddressModeClamp;

	check( hipBindTextureToArray(tex_voxels, d_volumeArray, channelDesc), "hipBindTextureToArray" );
	
	// initialize ray array
 	glm::vec3* d_rays;
	check( hipMalloc(&d_rays, sizeof(glm::vec3) * rays.size()), "hipMalloc" );
	check( hipMemcpy2D(d_rays, sizeof(glm::vec3), &rays[0].d, 2 * sizeof(glm::vec3), sizeof(glm::vec3), rays.size(), hipMemcpyHostToDevice), "hipMemcpy2D" );
	
	// initialize output arrays
	PointNormalPair* d_output;
	check( hipMalloc(&d_output, sizeof(PointNormalPair) * rays.size()), "hipMalloc" );

	// launch kernel
	dim3 block(256, 1, 1);
	dim3 grid((rays.size() + block.x - 1) / block.x, 1, 1);
	
	raymarchKernel<<<grid, block>>>(rays.front().o, d_rays, d_output, rays.size(), steps);
	check( hipDeviceSynchronize(), "kernel launch" );
	
	// retrieve intersection data
	thrust::device_ptr<PointNormalPair> out = thrust::device_pointer_cast(d_output);
	thrust::device_vector<PointNormalPair> d_results;
	d_results.resize(rays.size());
	
	thrust::device_vector<PointNormalPair>::iterator end;
	checkThrust( end = thrust::copy_if(out, out + rays.size(), d_results.begin(), IsNotZero()) , "copy_if" );
	results.resize(end - d_results.begin());
	checkThrust( thrust::copy(d_results.begin(), end, results.begin()), "copy" );
	
	// cleanup
	d_results.resize(0); d_results.shrink_to_fit();
	check( hipFree(d_rays), "hipFree");
	check( hipFree(d_output), "hipFree");
	check( hipFreeArray(d_volumeArray), "hipFreeArray" );
}
