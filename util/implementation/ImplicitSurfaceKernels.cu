#include "hip/hip_runtime.h"
/*
 * CUDA source file for ImplicitSurface class.
 */

#define check(expr, msg) { \
	hipError_t err; \
	err = expr; \
	if (err != hipSuccess) { std::cerr << "CUDA ERROR (" << hipGetErrorString(err) << "): " << msg << std::endl; } \
	}
	
#include <util/interface/ImplicitSurface.hpp>
#include <glm/interface/gtc/type_ptr.hpp>

#include <util/interface/cutil_math.h>

#include <limits.h>
#include <float.h>

// helper struct
typedef union {
	float3 vec;
	float elem[3];
} ufloat3;

// constant data
__constant__ ufloat3 c_bboxMin;
__constant__ ufloat3 c_bboxMax;
__constant__ int3 c_gridDims;

// textures
texture<float4, 3, hipReadModeElementType> tex_voxels;

__device__
bool clipRayAgainstAABB(ufloat3 org, ufloat3 dir, float& tnear, float& tfar) {
	ufloat3 T_1, T_2;
	double t_near = -FLT_MAX;
	double t_far = FLT_MAX;
	
	ufloat3 min = c_bboxMin, max = c_bboxMax;
	
	for (int i = 0; i < 3; i++){
		if (dir.elem[i] == 0){
			if ((org.elem[i] < min.elem[i]) || (org.elem[i] > max.elem[i])) {
				return false;
			}
		} else {
			T_1.elem[i] = (min.elem[i] - org.elem[i]) / dir.elem[i];
			T_2.elem[i] = (max.elem[i] - org.elem[i]) / dir.elem[i];

			if (T_1.elem[i] > T_2.elem[i]) {
				ufloat3 tmp = T_1;
				T_1 = T_2;
				T_2 = tmp;
			}
			if (T_1.elem[i] > t_near) {
				t_near = T_1.elem[i];
			}
			if (T_2.elem[i] < t_far) {
				t_far = T_2.elem[i];
			}
			if ( (t_near > t_far) || (t_far < 0.0f) || (fabsf(t_far - t_near) < 0.0001f) ) {
				return false;
			}
		}
	}
	
	tnear = t_near; tfar = t_far;
	
	return true;
}

__device__
void evaluateGrid(float3 target, float* value, float3* normal) {
	float3 coords = (target - c_bboxMin.vec) / (c_bboxMax.vec - c_bboxMin.vec);
	float4 interp = tex3D(tex_voxels, coords.x, coords.y, coords.z);
	if (value) {
		*value = interp.w;
	}
	if (normal) {
		*normal = make_float3(interp.x, interp.y, interp.z);
	}
}

__global__
void raymarchKernel(float3* d_rays, float3* d_output, unsigned n, unsigned steps) {
	unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (idx > n) {
		return;
	}
	
	ufloat3 org, dir;
	org.vec = d_rays[2*idx];
	dir.vec = d_rays[2*idx+1];
	
	float3 hit, normal;
	float tnear, tfar;
	
	bool intersects = clipRayAgainstAABB(org, dir, tnear, tfar);
	if (!intersects) {
		hit = make_float3(0.0f);
		normal = make_float3(0.0f);
	} else {
		tnear += 0.00001f;

		// march ray through surface
		float step = (tfar - tnear) / steps;

		// set sign to unknown at the beginning
		bool sign, signWasSet = false;

		// trace forward through the bounding box
		for (unsigned i = 1; i < steps-1; i++) { // -1 to not get too close to tfar
			float3 target = org.vec + dir.vec * (tnear + i * step);
			float implicitValue; float3 implicitNormal;
			evaluateGrid(target, &implicitValue, &implicitNormal);
			
			if (!signWasSet) {
				if (implicitValue < 0.0f) {
					sign = false;
					signWasSet = true;
				} else if (implicitValue > 0.0f) {
					sign = true;
					signWasSet = true;
				}
			} else {
				bool enteredSurface = (sign == true && implicitValue < 0.0f) || (sign == false && implicitValue > 0.0f);
				if (enteredSurface) { // ray entered surface
				
					// compute bounds between this and the last step for backward trace
					float tStart = tnear + (i-1) * step;
					float tEnd = tnear + i * step;
					float smallStep = (tEnd - tStart) / steps;
					
					// trace backwards from step that changed the sign
					for (int j = steps-1; j >= 0; j--) {
						target = org.vec + dir.vec * (tStart + j * smallStep);
						evaluateGrid(target, &implicitValue, &normal);
						
						bool exitedSurface = (sign == true && implicitValue > 0.0f) || (sign == false && implicitValue < 0.0f);
						if (exitedSurface) {
							hit = target;
							normal = implicitNormal;
							break;
						}
					}
					break;
				}
			}
		}
	}
	
	d_output[idx] = hit;
	d_output[n + idx] = normal;
}

void launchRaymarchKernel(const util::Grid3D& volume, const util::RayVector& rays, unsigned steps, PointNormalData& results) {
	
	// configure device
	check( hipDeviceSetCacheConfig( hipFuncCachePreferL1 ), "hipDeviceSetCacheConfig" );
	
	// copy bbox data to const
	check( hipMemcpyToSymbol(HIP_SYMBOL(c_bboxMin), glm::value_ptr(volume.bounds().min()), sizeof(float3)), "hipMemcpyToSymbol" );
	check( hipMemcpyToSymbol(HIP_SYMBOL(c_bboxMax), glm::value_ptr(volume.bounds().max()), sizeof(float3)), "hipMemcpyToSymbol" );
	check( hipMemcpyToSymbol(HIP_SYMBOL(c_gridDims), glm::value_ptr(volume.dimensions()), sizeof(int3)), "hipMemcpyToSymbol" );
	
	// initialize grid textures
	hipArray* d_volumeArray;
	Point3i dims = volume.dimensions();
	const hipExtent volumeSize = make_hipExtent(dims.x, dims.y, dims.z);
	
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
	check( hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize), "hipMalloc3DArray" );

	float4* h_volume = new float4[volume.size()];
	for (unsigned z = 0; z < dims.z; z++) {
		for (unsigned y = 0; y < dims.y; y++) {
			for (unsigned x = 0; x < dims.x; x++) {
				const Point3f& normal = volume.normal(x,y,z);
				h_volume[z * (dims.x * dims.y) + y * (dims.x) + x] = make_float4(normal.x, normal.y, normal.z, volume.value(x,y,z));
			}
		}
	}
	
	hipMemcpy3DParms copyParams = {0};
	copyParams.srcPtr   = make_hipPitchedPtr((void*)h_volume, volumeSize.width*sizeof(float4), volumeSize.width, volumeSize.height);
	copyParams.dstArray = d_volumeArray;
	copyParams.extent   = volumeSize;
	copyParams.kind     = hipMemcpyHostToDevice;
	check( hipMemcpy3D(&copyParams), "hipMemcpy3D" );

	tex_voxels.normalized = true;
	tex_voxels.filterMode = hipFilterModeLinear;
	tex_voxels.addressMode[0] = hipAddressModeClamp;
	tex_voxels.addressMode[1] = hipAddressModeClamp;
	tex_voxels.addressMode[2] = hipAddressModeClamp;

	check( hipBindTextureToArray(tex_voxels, d_volumeArray, channelDesc), "hipBindTextureToArray" );
	
	// initialize ray array
	float3* d_rays;
	check( hipMalloc(&d_rays, sizeof(float3) * 2 * rays.size()), "hipMalloc" );
	check( hipMemcpy(d_rays, rays.data(), sizeof(float3) * 2 * rays.size(), hipMemcpyHostToDevice), "hipMemcpy" );
	
	// initialize output array
	float3* d_output;
	check( hipMalloc(&d_output, sizeof(float3) * 2 * rays.size()), "hipMalloc" );
	
	// launch kernel
	dim3 block(256, 1, 1);
	dim3 grid((rays.size() + block.x - 1) / block.x, 1, 1);
	
	raymarchKernel<<<grid, block>>>(d_rays, d_output, rays.size(), steps);
	check( hipDeviceSynchronize(), "kernel launch" );
	
	// retrieve intersection data
	float3* h_output = new float3[2 * rays.size()];
	check( hipMemcpy(h_output, d_output, sizeof(float3) * 2 * rays.size(), hipMemcpyDeviceToHost), "hipMemcpy" );
	
	results.reserve(rays.size());
	for (unsigned i = 0; i < rays.size(); i++) {
		const float3& p = h_output[i];
		const float3& n = h_output[i + rays.size()];
		if (!(p.x == 0.0f && p.y == 0.0f && p.z == 0.0f)) {
			results.push_back( std::make_pair(Point3f(p.x, p.y, p.z), Point3f(n.x, n.y, n.z)) );
		}
	}
	
	// cleanup
	delete[] h_volume;
	delete[] h_output;
	check( hipFree(d_rays), "hipFree");
	check( hipFree(d_output), "hipFree");
}
